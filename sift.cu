#include "hip/hip_runtime.h"
/**
 * SIFT implementation.
 */

//FIXME: in extract, return sift points.
//       add cuda, write report, test.

#include "sift.hh"

Sift::Sift (const IplImage& src_, double pt, double te, double nt,
            int O_, int S_, int o_min_)
  : n_keys (0),
    n_keys_res (0),
    keys (0),
    src (src_),
    peak_threshold (pt),
    edge_threshold (te),
    norm_threshold (nt),
    O ((O < 0) ? compute_o_min (o_min_, src_.width, src_.height) : O_),
    S (S_),
    o_min (o_min_),
    s_min (-1),
    s_max (S_ + 1),
    w (src.width),
    h (src.height),
    s (shift_left (w, -o_min) * shift_left (h, -o_min) * sizeof (double)),
    xo (1),
    yo (w),
    so (w * h),
    sigmak_ (pow(2.0, 1.0 / S_)), // 2^(1/S)
    sigman_ (0.5),
    sigma0_ (1.6 * sigmak_),
    dsigma0_ (sigma0_ * sqrt (1.0 - 1.0/(sigmak_*sigmak_))), // sigma0 * sqrt(1 - 1/sigmak²)
    oCur_ (o_min_),
    oW_ (O),
    oH_ (0),
    oGrad_ (o_min_ - 1),
    octave_ (h_malloc<double> (s*(s_max-s_min+1))),
    dog_ (h_malloc<double> (s*(s_max-s_min))),
    gradient_ (h_malloc<double> (s*2*(s_max-s_min))),
    tmp_ (h_malloc<double> (s)),
    im_ (h_malloc<double> (src_.width * src.height * sizeof (double)))
{
  std::cout << "Create SIFT filter with parameter:" << std::endl
            << "* Number of octaves: " << O_ << std::endl
            << "* Min octave: " << o_min_ << std::endl
            << "* S: " << S_ << std::endl
            << "* Peak/Edge/Norm thresholds: " << pt << "/" << te << "/" << nt
            << std::endl;

  // Convert openCV image to double*
  int offset = 0;
  for (int y = 0; y < src.height; ++y)
    for (int x = 0; x < src.width; ++x)
      im_[offset++] = cvGet2D(&src, y, x).val[0];
}

Sift::~Sift ()
{
  free (keys);
  free (im_);
  free (tmp_);
  free (dog_);
  free (gradient_);
  free (octave_);
}

bool
Sift::process ()
{
  std::cout << "+Process" << std::endl;
  if (!O)
    return false;
  int o, s;
  double sa, sb;

  oCur_ = o_min;
  n_keys = 0;
  oW_ = shift_left (src.width,  -oCur_);
  oH_ = shift_left (src.height, -oCur_);
  s = oW_ * oH_;

  double* octave = get_octave (s_min);

  if (o_min < 0)
    {
      /* double once */
      copy_and_upsample_rows (tmp_, im_, w, h);
      copy_and_upsample_rows (octave, tmp_, h, 2 * w);

      /* double more */
      for (o = -1; o > o_min; --o)
        {
          copy_and_upsample_rows (tmp_, octave,
                                    w << -o,      h << -o );
          copy_and_upsample_rows (octave, tmp_,
                                  w << -o, 2 * (h << -o));
        }
    }
  else if (o_min > 0)
    copy_and_downsample (octave, im_, w, h, o_min);
  else
    memcpy(octave, im_, s);

  // adjust smoothing.
  sa = sigma0_ * pow (sigmak_, s_min);
  sb = sigman_ * pow (2.0, -o_min);

  if (sa > sb) {
    double sd = sqrt (sa*sa - sb*sb);
    imsmooth (octave, tmp_, octave, oW_, oH_, sd);
  }

  // compute octave.
  for(s = s_min + 1; s <= s_max; ++s) {
    double sd = dsigma0_ * pow (sigmak_, s);
    imsmooth (get_octave(s), tmp_,
              get_octave(s - 1), w, h, sd);
  }
  std::cout << "-Process" << std::endl;
  return true;
}

bool
Sift::process_next ()
{
  std::cout << "+Process next" << std::endl;
  if (oCur_ == o_min + O - 1)
    return false;

  int s_best = min (s_min + S, s_max);
  double* pt = get_octave (s_best);
  double* octave = get_octave (s_min);

  copy_and_downsample (octave, pt, w, h, 1);

  oCur_ += 1, n_keys = 0;
  oW_ = shift_left (w,  - oCur_);
  oH_ = shift_left (h, - oCur_);

  double sa = sigma0_ * powf (sigmak_, s_min);
  double sb = sigma0_ * powf (sigmak_, s_best - S);

  if (sa > sb)
    {
      double sd = sqrt (sa*sa - sb*sb);
      imsmooth (octave, tmp_, octave, oW_, oH_, sd);
    }
  std::cout << "-Process next" << std::endl;
  return true;
}

#define SINGLE_EPSILON 1.19209290E-07F
double
Sift::normalize_histogram (double *begin, double *end)
{
  double norm = 0.0;
  for (double* iter = begin ; iter != end ; ++iter)
    norm += (*iter) * (*iter);
  norm = sqrt (norm) + SINGLE_EPSILON;
  for (double* iter = begin; iter != end ; ++iter)
    *iter /= norm;
  return norm;
}
#undef SINGLE_EPSILON

#define atd(dbinx,dbiny,dbint) *(dpt + (dbint)*binto + (dbiny)*binyo + (dbinx)*binxo)
void
Sift::compute_keypoint_descriptor(double descr[128], int ind, double angle)
{
  assert (ind < n_keys);
  SiftKeypoint* k = keys + ind;

  const double magnif      = 3.0;
  const int    NBO         = 8;
  const int    NBP         = 4;
  double       xper        = pow (2.0, oCur_);
  const int    xo          = 2;         /* x-stride */
  const int    yo          = 2 * w;     /* y-stride */
  const int    so          = 2 * w * h; /* s-stride */
  double       x           = k->x     / xper;
  double       y           = k->y     / xper;
  double       sigma       = k->sigma / xper;

  int          xi          = (int) (x + 0.5);
  int          yi          = (int) (y + 0.5);
  int          si          = k->is;

  const double st0         = sin (angle);
  const double ct0         = cos (angle);
  const double SBP         = magnif * sigma;
  const int W           = (int)floor
    (sqrt(2.0) * SBP * (NBP + 1) / 2.0 + 0.5);

  const int binto = 1;          /* bin theta-stride */
  const int binyo = NBO * NBP;  /* bin y-stride */
  const int binxo = NBO;        /* bin x-stride */

  /* check bounds */
  if(k->o  != oCur_ || xi <  0 || xi >= w || yi <  0 ||
     yi >= h - 1 || si < s_min + 1 || si > s_max - 2)
    return;

  /* synchronize gradient buffer */
  update_gradient ();

  /* clear descriptor */
  memset (descr, 0, sizeof(double) * NBO*NBP*NBP);

  /* Center the scale space and the descriptor on the current keypoint.
   * Note that dpt is pointing to the bin of center (SBP/2,SBP/2,0).
   */
  const double* pt  = gradient_ + xi*xo + yi*yo + (si - s_min - 1)*so;
  double* dpt = descr + (NBP/2) * binyo + (NBP/2) * binxo;

  /*
   * Process pixels in the intersection of the image rectangle
   * (1,1)-(M-1,N-1) and the keypoint bounding box.
   */
  for(int dyi =  max (- W, 1 - yi); dyi <= min (+ W, h - yi - 2); ++dyi)
    for(int dxi =  max (- W, 1 - xi); dxi <= min (+ W, w - xi - 2); ++ dxi)
      {
        /* retrieve */
        double mod   = *( pt + dxi*xo + dyi*yo + 0 );
        double angle_ = *( pt + dxi*xo + dyi*yo + 1 );
        double theta = mod_2pi (angle_ - angle);

        /* fractional displacement */
        double dx = xi + dxi - x;
        double dy = yi + dyi - y;

        /* get the displacement normalized w.r.t. the keypoint
           orientation and extension */
        double nx = ( ct0 * dx + st0 * dy) / SBP;
        double ny = (-st0 * dx + ct0 * dy) / SBP;
        double nt = NBO * theta / (2 * M_PI);

        /* Get the Gaussian weight of the sample. The Gaussian window
         * has a standard deviation equal to NBP/2. Note that dx and dy
         * are in the normalized frame, so that -NBP/2 <= dx <=
         * NBP/2. */
        double const wsigma = NBP/2;
        double win = fast_expn
          ((nx*nx + ny*ny)/(2.0 * wsigma * wsigma));

        /* The sample will be distributed in 8 adjacent bins.
           We start from the ``lower-left'' bin. */
        int binx = (int)floor (nx - 0.5);
        int biny = (int)floor (ny - 0.5);
        int bint = (int)floor (nt);
        double rbinx = nx - (binx + 0.5);
        double rbiny = ny - (biny + 0.5);
        double rbint = nt - bint;

        /* Distribute the current sample into the 8 adjacent bins*/
        for(int dbinx = 0; dbinx < 2; ++dbinx)
          for(int dbiny = 0; dbiny < 2; ++dbiny)
            for(int dbint = 0; dbint < 2; ++dbint)
              {
                if (binx + dbinx >= - (NBP/2) &&
                    binx + dbinx <    (NBP/2) &&
                    biny + dbiny >= - (NBP/2) &&
                    biny + dbiny <    (NBP/2) )
                  {
                    double weight = win
                      * mod
                      * abs (1 - dbinx - rbinx)
                      * abs (1 - dbiny - rbiny)
                      * abs (1 - dbint - rbint);
                    atd(binx+dbinx, biny+dbiny, (bint+dbint) % NBO) += weight;
                  }
              }
      }

  /* Normalize the histogram to L2 unit length. */
  double norm = normalize_histogram (descr, descr + NBO*NBP*NBP);

  /* Set the descriptor to zero if it is lower than our norm_threshold */
  if(norm_threshold && norm <  norm_threshold)
    for (int bin = 0; bin < NBO*NBP*NBP; ++ bin)
      descr [bin] = 0;
  else
    {
      /* Truncate at 0.2. */
      for(int bin = 0; bin < NBO*NBP*NBP; ++ bin)
        if (descr [bin] > 0.2) descr [bin] = 0.2;

      /* Normalize again. */
      normalize_histogram (descr, descr + NBO*NBP*NBP);
    }
}

Sift::features_t
Sift::extract ()
{
  features_t res;

  if (!process ())
    return res;
  while (true)
    {
      detect ();

      for (int i=0; i<n_keys; ++i)
        {
          double angles[4];
          int n_angles = compute_keypoint_orientation (i, angles);

          for (int q=0; q < n_angles; ++q)
            {
              double descr[128];
              compute_keypoint_descriptor(descr, i, angles [q]);

              SiftFeaturePoint fp;
              fp.x = keys[i].x, fp.y = keys[i].y;
              fp.scale = keys[i].sigma;
              fp.angle = angles[q];
              for (int k = 0; k < 128; ++k)
                fp.desc[k] = 512 * descr[k];
              res.push_back (fp);
            }
        }

      // build SIFT descriptors.
      if (!process_next ())
        return res;
    }
}

void
Sift::compute_dog ()
{
  double* pt = dog_;
  for (int s = s_min; s <= s_max - 1; ++s)
    {
      double* src_a = get_octave (s);
      double* src_b = get_octave (s + 1);
      double* end_a = src_a + oW_ * oH_;
      while (src_a != end_a)
        *pt++ = *src_b++ - *src_a++;
    }
}




#define CHECK_NEIGHBORS(CMP,SGN)                \
  ( v CMP ## = SGN 0.8 * peak_threshold &&      \
    v CMP *(pt + xo) &&                         \
    v CMP *(pt - xo) &&                         \
    v CMP *(pt + so) &&                         \
    v CMP *(pt - so) &&                         \
    v CMP *(pt + yo) &&                         \
    v CMP *(pt - yo) &&                         \
                                                \
    v CMP *(pt + yo + xo) &&                    \
    v CMP *(pt + yo - xo) &&                    \
    v CMP *(pt - yo + xo) &&                    \
    v CMP *(pt - yo - xo) &&                    \
                                                \
    v CMP *(pt + xo      + so) &&               \
    v CMP *(pt - xo      + so) &&               \
    v CMP *(pt + yo      + so) &&               \
    v CMP *(pt - yo      + so) &&               \
    v CMP *(pt + yo + xo + so) &&               \
    v CMP *(pt + yo - xo + so) &&               \
    v CMP *(pt - yo + xo + so) &&               \
    v CMP *(pt - yo - xo + so) &&               \
                                                \
    v CMP *(pt + xo      - so) &&               \
    v CMP *(pt - xo      - so) &&               \
    v CMP *(pt + yo      - so) &&               \
    v CMP *(pt - yo      - so) &&               \
    v CMP *(pt + yo + xo - so) &&               \
    v CMP *(pt + yo - xo - so) &&               \
    v CMP *(pt - yo + xo - so) &&               \
    v CMP *(pt - yo - xo - so) )

void
Sift::detect_maxima ()
{
  std::cout << "+Detect maxima" << std::endl;
  double* pt  = dog_ + xo + yo + so;

  for (int s = s_min + 1; s <= s_max - 2; ++s)
    {
      for(int y = 1; y < h - 1; ++y)
        {
          for(int x = 1; x < w - 1; ++x)
            {
              double v = *pt;
              if (CHECK_NEIGHBORS(>,+) ||
                  CHECK_NEIGHBORS(<,-) )
                {
                  /* make room for more keypoints */
                  if (n_keys >= n_keys_res)
                    {
                      n_keys_res += 500;
                      if (keys)
                        keys = (SiftKeypoint*) realloc (keys,
                                                        n_keys_res *
                                                        sizeof(SiftKeypoint));
                      else
                        keys = h_malloc<SiftKeypoint> (n_keys_res *
                                                       sizeof(SiftKeypoint));
                    }

                  SiftKeypoint* k = keys + (n_keys ++);
                  k->ix = x;
                  k->iy = y;
                  k->is = s;
                }
              pt += 1;
            }
          pt += 2;
        }
      pt += 2 * yo;
    }
  std::cout << "-Detect maxima" << std::endl;
}

#define at(dx,dy,ds) (*( pt + (dx)*xo + (dy)*yo + (ds)*so))
#define Aat(i,j)     (A[(i)+(j)*3])
void
Sift::refine_maxima ()
{
  std::cout << "+Refine maxima" << std::endl;
  SiftKeypoint* k = keys;
  int i = 0, j = 0, jj = 0, ii = 0;

  double maxa = 0;
  double maxabsa = 0;
  int maxi  = -1;
  double tmp = 0.;
  double* pt = 0;
  double xper  = pow (2.0, oCur_);

  for (i = 0; i < n_keys; ++i)
    {
      int x =  keys[i].ix;
      int y =  keys[i].iy;
      int s =  keys[i].is;

      double Dx=0, Dy=0, Ds=0,
        Dxx=0, Dyy=0, Dss=0,
        Dxy=0, Dxs=0, Dys=0;
      double A [3*3], b [3];
      int dx = 0, dy = 0;

      for (int iter = 0; iter < 5; ++iter)
        {
          x += dx;
          y += dy;

          pt = dog_ + xo * x + yo * y + so * (s - s_min);

          Dx = 0.5 * (at(+1,0,0) - at(-1,0,0));
          Dy = 0.5 * (at(0,+1,0) - at(0,-1,0));
          Ds = 0.5 * (at(0,0,+1) - at(0,0,-1));

          Dxx = (at(+1,0,0) + at(-1,0,0) - 2.0 * at(0,0,0));
          Dyy = (at(0,+1,0) + at(0,-1,0) - 2.0 * at(0,0,0));
          Dss = (at(0,0,+1) + at(0,0,-1) - 2.0 * at(0,0,0));

          Dxy = 0.25 * ( at(+1,+1,0) + at(-1,-1,0) - at(-1,+1,0) - at(+1,-1,0) );
          Dxs = 0.25 * ( at(+1,0,+1) + at(-1,0,-1) - at(-1,0,+1) - at(+1,0,-1) );
          Dys = 0.25 * ( at(0,+1,+1) + at(0,-1,-1) - at(0,-1,+1) - at(0,+1,-1) );

          Aat(0,0) = Dxx, Aat(1,1) = Dyy, Aat(2,2) = Dss,
            Aat(0,1) = Aat(1,0) = Dxy, Aat(0,2) = Aat(2,0) = Dxs,
            Aat(1,2) = Aat(2,1) = Dys;

          b[0] = - Dx, b[1] = - Dy, b[2] = - Ds;

          /* Gauss elimination */
          for(j = 0; j < 3; ++j)
            {
              /* look for the maximally stable pivot */
              for (int i = j; i < 3; ++i)
                {
                  double a = Aat (i,j), absa = abs (a);
                  if (absa > maxabsa)
                    maxa = a, maxabsa = absa, maxi = i;
                }
            }

          /* if singular give up */
          if (maxabsa < 1e-10f)
            {
              b[0] = 0, b[1] = 0, b[2] = 0;
              break;
            }

          i = maxi;

          /* swap j-th row with i-th row and normalize j-th row */
          for(jj = j; jj < 3; ++jj)
            tmp = Aat(i,jj); Aat(i,jj) = Aat(j,jj); Aat(j,jj) = tmp,
                                                      Aat(j,jj) /= maxa;
          tmp = b[j]; b[j] = b[i]; b[i] = tmp;
          b[j] /= maxa;

          /* elimination */
          for (ii = j+1; ii < 3; ++ii)
            {
              double x = Aat(ii,j);
              for (jj = j; jj < 3; ++jj)
                Aat(ii,jj) -= x * Aat(j,jj);
              b[ii] -= x * b[j];
            }

          /* backward substitution */
          for (i = 2; i > 0; --i)
            {
              double x = b[i];
              for (ii = i-1; ii >= 0; --ii)
                b[ii] -= x * Aat(ii,i);
            }

          /* ........................................................... */
          /* If the translation of the keypoint is big, move the keypoint
           * and re-iterate the computation. Otherwise we are all set.
           */

          dx= ((b[0] >  0.6 && x < w - 2) ?  1 : 0)
            + ((b[0] < -0.6 && x > 1    ) ? -1 : 0);

          dy= ((b[1] >  0.6 && y < h - 2) ?  1 : 0)
            + ((b[1] < -0.6 && y > 1    ) ? -1 : 0);

          if (dx == 0 && dy == 0) break;
        }

      /* check threshold and other conditions */
      {
        double val   = at(0,0,0)
          + 0.5 * (Dx * b[0] + Dy * b[1] + Ds * b[2]);
        double score = (Dxx+Dyy)*(Dxx+Dyy) / (Dxx*Dyy - Dxy*Dxy);
        double xn = x + b[0];
        double yn = y + b[1];
        double sn = s + b[2];

        bool good =
          abs (val)  > peak_threshold           &&
          score           < (edge_threshold+1)*(edge_threshold+1)
          /edge_threshold    &&
          score           >= 0                  &&
          abs (b[0]) <  1.5                     &&
          abs (b[1]) <  1.5                     &&
          abs (b[2]) <  1.5                     &&
          xn              >= 0                  &&
          xn              <= w - 1              &&
          yn              >= 0                  &&
          yn              <= h - 1              &&
          sn              >= s_min              &&
          sn              <= s_max;

        if (good) {
          k->o = oCur_;
          k->ix = x, k->iy = y, k->is = s;
          k->s = sn;
          k->x = xn * xper, k->y = yn * xper;
          k->sigma = sigma0_ * pow (2.0, sn/S) * xper;
          ++k;
        }

      } /* done checking */
    } /* next keypoint to refine */

  /* update keypoint count */
  n_keys = k - keys;
  std::cout << "-Refine maxima" << std::endl;
}
#undef at

#define SAVE_BACK                                       \
  *gradient_++ = sqrt (gx*gx + gy*gy);                  \
  *gradient_++ = mod_2pi(atan2 (gy, gx) + 2*M_PI);      \
  ++src


void
Sift::update_gradient ()
{
  if (oGrad_ == oCur_)
    return;

  for (int s  = s_min + 1;
       s <= s_max - 2; ++ s)
    {
      double* src;
      double* end;
      double gx, gy;
      gradient_ = gradient_ + 2 * so * (s - s_min -1);
      src  = get_octave (s);

      /* first first row */
      gx = src[+xo] - src[0];
      gy = src[+yo] - src[0];
      SAVE_BACK;

      /* middle first row */
      end = (src - 1) + w - 1;
      while (src < end)
        {
          gx = 0.5 * (src[+xo] - src[-xo]);
          gy =        src[+yo] - src[0];
          SAVE_BACK;
        }

      /* first first row */
      gx = src[0]   - src[-xo];
      gy = src[+yo] - src[0];
      SAVE_BACK;

      for (int y = 1; y < h -1; ++y)
        {
          /* first middle row */
          gx =        src[+xo] - src[0];
          gy = 0.5 * (src[+yo] - src[-yo]);
          SAVE_BACK;

          /* middle middle row */
          end = (src - 1) + w - 1;
          while (src < end)
            {
              gx = 0.5 * (src[+xo] - src[-xo]);
              gy = 0.5 * (src[+yo] - src[-yo]);
              SAVE_BACK;
            }

          /* last middle row */
          gx =        src[0]   - src[-xo];
          gy = 0.5 * (src[+yo] - src[-yo]);
          SAVE_BACK;
        }

      /* first last row */
      gx = src[+xo] - src[0];
      gy = src[  0] - src[-yo];
      SAVE_BACK;

      /* middle last row */
      end = (src - 1) + w - 1;
      while (src < end)
        {
          gx = 0.5 * (src[+xo] - src[-xo]);
          gy =        src[0]   - src[-yo];
          SAVE_BACK;
        }

      /* last last row */
      gx = src[0]   - src[-xo];
      gy = src[0]   - src[-yo];
      SAVE_BACK;
    }
  oGrad_ = oCur_;
}

#define at(dx,dy) (*(pt + xo * (dx) + yo * (dy)))
int
Sift::compute_keypoint_orientation (int ind, double angles [4])
{
  assert (ind < n_keys);
  SiftKeypoint* k = keys + ind;

  const double winf = 1.5;
  double xper = pow (2.0, oCur_);

  const int xo = 2;         /* x-stride */
  const int yo = 2 * w;     /* y-stride */
  const int so = 2 * w * h; /* s-stride */
  double x = k->x/xper, y = k->y/xper;
  double sigma = k->sigma/xper;

  int xi = (int) (x + 0.5),
    yi = (int) (y + 0.5), si = k->is;

  const double sigmaw = winf * sigma;
  int W = (int)max (floor (3.0 * sigmaw), 1.);
  int nangles = 0;
  enum { nbins = 36 };

  double hist [nbins], maxh;

  /* skip if the keypoint octave is not current */
  if(k->o != oCur_)
    return 0;

  /* skip the keypoint if it is out of bounds */
  if(xi < 0 || xi > w - 1 || yi < 0 || yi > h - 1 ||
     si < s_min + 1 || si > s_max - 2 )
    return 0;

  /* make gradient up to date */
  update_gradient ();

  /* clear histogram */
  memset (hist, 0, sizeof(double) * nbins);

  /* compute orientation histogram */
  double* pt =  gradient_ + xo*xi + yo*yi + so*(si - s_min - 1);

  for (int ys  =  max (- W, - yi);
       ys <=  min (+ W, h - 1 - yi); ++ys)
    for (int xs  = max (- W,       - xi);
         xs <= min (+ W, w - 1 - xi); ++xs)
      {
        double dx = (double)(xi + xs) - x;
        double dy = (double)(yi + ys) - y;
        double r2 = dx*dx + dy*dy;
        double wgt, mod, ang, fbin;

        /* limit to a circular window */
        if (r2 >= W*W + 0.6)
          continue;

        wgt  = fast_expn (r2 / (2*sigmaw*sigmaw));
        mod  = *(pt + xs*xo + ys*yo    );
        ang  = *(pt + xs*xo + ys*yo + 1);
        fbin = nbins * ang / (2 * M_PI);

        int    bin  = (int)floor (fbin - 0.5);
        double rbin = fbin - bin - 0.5;
        hist [(bin + nbins) % nbins] += (1 - rbin) * mod * wgt;
        hist [(bin + 1    ) % nbins] += (rbin) * mod * wgt;
      }

  /* smooth histogram */
  for (int iter = 0; iter < 6; iter++)
    {
      double prev  = hist [nbins - 1];
      double first = hist [0];
      int i;
      for (i = 0; i < nbins - 1; i++)
        {
          double newh = (prev + hist[i] + hist[(i+1) % nbins]) / 3.0;
          prev = hist[i];
          hist[i] = newh;
        }
      hist[i] = (prev + hist[i] + first) / 3.0;
    }

  /* find the histogram maximum */
  maxh = 0;
  for (int i = 0; i < nbins; ++i)
    maxh = max (maxh, hist [i]);

  /* find peaks within 80% from max */
  int n_angles = 0;
  for (int i = 0; i < nbins; ++i)
    {
      double h0 = hist [i];
      double hm = hist [(i - 1 + nbins) % nbins];
      double hp = hist [(i + 1 + nbins) % nbins];

      /* is this a peak? */
      if (h0 > 0.8*maxh && h0 > hm && h0 > hp)
        {
          /* quadratic interpolation */
          double di = - 0.5 * (hp - hm) / (hp + hm - 2 * h0);
          double th = 2 * M_PI * (i + di + 0.5) / nbins;
          angles [ nangles++ ] = th;
          if( nangles == 4 )
            return n_angles;
        }
    }
  return n_angles;
}
#undef at

void
Sift::copy_and_upsample_rows (double* dst,
                             double const* src,
                             int width, int height)
{
  std::cout << "+copy_and_upsample_rows" << std::endl;
  int x, y;
  double a, b;

  for (y = 0; y < height; ++y)
    {
      b = a = *src++;
      for (x = 0; x < width - 1; ++x)
        {
          b = *src++;
          *dst = a; dst += height;
          *dst = 0.5 * (a + b); dst += height;
          a = b;
        }
      *dst = b; dst += height;
      *dst = b; dst += height;
      dst += 1 - width * 2 * height;
    }
  std::cout << "-copy_and_upsample_rows" << std::endl;
}

void
Sift::copy_and_downsample (double* dst,  double const* src,
                           int width, int height, int d)
{
  std::cout << "+copy_and_downsample" << std::endl;
  int x, y;

  d = 1 << d; /* d = 2^d */
  for (y = 0; y < height; y+=d)
    {
      double const* srcrowp = src + y * width;
      for(x = 0; x < width - (d-1); x+=d)
        {
          *dst++ = *srcrowp;
          srcrowp += d;
        }
    }
  std::cout << "-copy_and_downsample" << std::endl;
}

void
Sift::convtransp (double* dst,
            double const* src,
            double const* filt,
            int width, int height, int filt_width)
{
  std::cout << "+convtransp " << width << "/" << height << "/" << filt_width << std::endl;
  int i, j;

  for(j = 0; j < height; ++j) {
    for(i = 0; i < width; ++i) {
      double        acc   = 0.0;
      double const *g     = filt;
      double const *start = src + (i - filt_width);
      double const *stop;
      double        x;

      /* beginning */
      //std::cout << "+b" << std::endl;
      stop = src + max (0, i - filt_width);
      x    = *stop;
      while (start <= stop) { acc += (*g++) * x; start++; }
      //std::cout << "-b" << std::endl;

      /* middle */
      //std::cout << "+m" << std::endl;
      stop =  src + min (width - 1, i + filt_width);
      while (start <  stop) acc += (*g++) * (*start++);
      //std::cout << "-m" << std::endl;

      /* end */
      //std::cout << "+e" << std::endl;
      x  = *start;
      stop = src + (i + filt_width);
      while (start <= stop) { acc += (*g++) * x; start++; }
      //std::cout << "-e" << std::endl;

      /* save */
      *dst = acc; //FIXME??
      dst += height;

      assert (g - filt == 2 * filt_width +1);
    }
    /* next column */
    src += width;
    dst -= width*height - 1;
  }
  std::cout << "-convtransp" << std::endl;
}

void
Sift::imsmooth(double* dst,
         double* temp,
         double  const* src,
         int width, int height, double sigma)
{
  std::cout << "+imsmooth " << width << "/" << height << "/" << sigma << std::endl;
  static double   *filt       = 0;
  static int    filt_width = -1;
  static double filt_static_sigma = -1.0;
  enum          { filt_static_res = 1024 };
  static double    filt_static [2 * filt_static_res + 1];

  int j;

  if (sigma < (double)(1e-5))
    {
      dst = (double*) memcpy (dst,src,width*height*sizeof(double));
      return;
    }

  /* window width */
  filt_width = (int) ceil (4.0 * sigma);

  /* setup filter only if not available from previous iteration*/
  if (filt_static_sigma != sigma)
    {
      double acc = 0.0;

      if (filt_width <= filt_static_res)
        {
          /* use static buffer */
          filt = filt_static;
          filt_static_sigma = sigma;
        } else {
        /* dynamically allocate a larger buffer */
        filt = h_malloc<double>  (sizeof(double) * (2*filt_width+1));
      }

      for (j = 0; j < 2 * filt_width + 1; ++j) {
        double  d = (double)(j - filt_width) / (double)(sigma);
        filt [j] = exp (- 0.5 * d * d);
        acc += filt [j];
      }

      /* normalize */
      for (j = 0; j < 2 * filt_width + 1; ++j)
        filt [j] /= acc;
    }

  /* convolve */
  convtransp (temp, src, filt,
              width, height, filt_width);
  convtransp (dst, temp, filt,
              height, width, filt_width);
  /* free buffer? */
  if (filt_static_sigma != sigma)
    free (filt);
  std::cout << "-imsmooth" << std::endl;
}
